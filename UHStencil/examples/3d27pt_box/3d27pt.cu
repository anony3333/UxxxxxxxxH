// to run:
// nvcc -c -O3 -o 3d27pt_cu.o 3d27pt.cu && nvcc -ccbin clang 3d27pt_main.cpp 3d27pt_cu.o -lcudart -lcuda -o && ./a.out

#include <hip/hip_runtime.h>
#include <stdio.h>
// dsl 输入
#define M 258  
#define N 258
#define K 258
#define Halo 1

// 编译时输入（选项/uh文件）
#define gridSize_x 8 
#define gridSize_y 8
#define gridSize_z 8
#define blockSize_x 4
#define blockSize_y 4
#define blockSize_z 4

__global__ void stencil_3d27pt_kernel_1(double* input, double* output) 
{
    // u_int tx = threadIdx.x;
    // u_int ty = threadIdx.y;
    // u_int tz = threadIdx.z;
    // u_int bx = blockIdx.x;
    // u_int by = blockIdx.y;
    // u_int bz = blockIdx.z;

    // 编译时推断
    const u_int points_per_thread_x = (M - 2 * Halo) / gridSize_x / blockSize_x; 
    const u_int points_per_thread_y = (N - 2 * Halo) / gridSize_y / blockSize_y;
    const u_int points_per_thread_z = (K - 2 * Halo) / gridSize_z / blockSize_z;

    // 编译时确定
    const u_int points_per_block_x = points_per_thread_x * blockSize_x; 
    const u_int points_per_block_y = points_per_thread_y * blockSize_y;
    const u_int points_per_block_z = points_per_thread_z * blockSize_z;

    // inner tile 编译时指定
    const u_int tile_size_x = 2;
    const u_int tile_size_y = 2;
    const u_int tile_size_z = 2;
    // shared mem 
    __shared__ double in_shm[tile_size_z * blockSize_z + 2 * Halo]
                            [tile_size_y * blockSize_y + 2 * Halo]
                            [tile_size_x * blockSize_x + 2 * Halo];
    __shared__ double out_shm[tile_size_z * blockSize_z]
                            [tile_size_y * blockSize_y]
                            [tile_size_x * blockSize_x];

    double (*in)[N][K] = (double(*)[N][K])input;
    double (*out)[N][K] = (double(*)[N][K])output;

    // 编译时推断
    const u_int num_tile_z = points_per_thread_z / tile_size_z;
    const u_int num_tile_y = points_per_thread_y / tile_size_y;
    const u_int num_tile_x = points_per_thread_x / tile_size_x;

    for (u_int i_tile_z = 0; i_tile_z < num_tile_z; i_tile_z ++) {
    for (u_int i_tile_y = 0; i_tile_y < num_tile_y; i_tile_y ++) {
    for (u_int i_tile_x = 0; i_tile_x < num_tile_x; i_tile_x ++) {

        u_int iz_st = blockIdx.z * points_per_block_z + i_tile_z * tile_size_z * blockSize_z;
        u_int iy_st = blockIdx.y * points_per_block_y + i_tile_y * tile_size_y * blockSize_y;
        u_int ix_st = blockIdx.x * points_per_block_x + i_tile_x * tile_size_x * blockSize_x;

        // load shared mem
        for (u_int iz = threadIdx.z; iz < blockSize_z * tile_size_z + 2 * Halo; iz += blockSize_z) {
            for (u_int iy = threadIdx.y; iy < blockSize_y * tile_size_y + 2 * Halo; iy += blockSize_y) {
                for (u_int ix = threadIdx.x; ix < blockSize_x * tile_size_x + 2 * Halo; ix += blockSize_x) {
                    in_shm[iz][iy][ix] = in[iz_st + iz][iy_st + iy][ix_st + ix];
                }
            }
        }
        // in_shm[0][0][0] = in[1][1][1];
        __syncthreads();
        // compute
        for (u_int iz = 0; iz < tile_size_z; iz ++) {
            for (u_int iy = 0; iy < tile_size_y; iy ++) {
                for (u_int ix = 0; ix < tile_size_x; ix ++) {
                    u_int zz = iz * blockSize_z + threadIdx.z; 
                    u_int yy = iy * blockSize_y + threadIdx.y; 
                    u_int xx = ix * blockSize_x + threadIdx.x; 
                    out_shm[zz][yy][xx] = 0.1*in_shm[zz][yy][xx] 
                                        + 0.2*in_shm[zz][yy][xx+Halo] 
                                        + 0.3*in_shm[zz][yy][xx+2*Halo]
                                        + 0.4*in_shm[zz][yy+Halo][xx]
                                        + 0.5*in_shm[zz][yy+Halo][xx+Halo]
                                        + 0.6*in_shm[zz][yy+Halo][xx+2*Halo]
                                        + 0.7*in_shm[zz][yy+2*Halo][xx] 
                                        + 0.8*in_shm[zz][yy+2*Halo][xx+Halo] 
                                        + 0.9*in_shm[zz][yy+2*Halo][xx+2*Halo]
                                        // panel 1
                                        + 0.1*in_shm[zz+Halo][yy][xx] 
                                        + 0.2*in_shm[zz+Halo][yy][xx+Halo] 
                                        + 0.3*in_shm[zz+Halo][yy][xx+2*Halo]
                                        + 0.4*in_shm[zz+Halo][yy+Halo][xx] 
                                        + 0.5*in_shm[zz+Halo][yy+Halo][xx+Halo] 
                                        + 0.6*in_shm[zz+Halo][yy+Halo][xx+2*Halo]
                                        + 0.7*in_shm[zz+Halo][yy+2*Halo][xx] 
                                        + 0.8*in_shm[zz+Halo][yy+2*Halo][xx+Halo] 
                                        + 0.9*in_shm[zz+Halo][yy+2*Halo][xx+2*Halo]
                                        // panel 2
                                        + 0.1*in_shm[zz+2*Halo][yy][xx] 
                                        + 0.2*in_shm[zz+2*Halo][yy][xx+Halo] 
                                        + 0.3*in_shm[zz+2*Halo][yy][xx+2*Halo]
                                        + 0.4*in_shm[zz+2*Halo][yy+Halo][xx] 
                                        + 0.5*in_shm[zz+2*Halo][yy+Halo][xx+Halo] 
                                        + 0.6*in_shm[zz+2*Halo][yy+Halo][xx+2*Halo]
                                        + 0.7*in_shm[zz+2*Halo][yy+2*Halo][xx] 
                                        + 0.8*in_shm[zz+2*Halo][yy+2*Halo][xx+Halo] 
                                        + 0.9*in_shm[zz+2*Halo][yy+2*Halo][xx+2*Halo];
                }
            }
        }
        __syncthreads();
        // store
        for (u_int iz = threadIdx.z; iz < tile_size_z * blockSize_z; iz += blockSize_z) {
            for (u_int iy = threadIdx.y; iy < tile_size_y * blockSize_y; iy += blockSize_y) {
                for (u_int ix = threadIdx.x; ix < tile_size_x * blockSize_x; ix += blockSize_x) {
                    out[iz_st + Halo + iz ][iy_st + Halo + iy][ix_st + Halo + ix] = out_shm[iz][iy][ix];
                }
            }
        }
        // out[1][1][1] = in_shm[0][0][0];
    }
    }
    }
}

void stencil_3d27pt_box_cuda(double* input, double* output) {
    dim3 grid{gridSize_x, gridSize_y, gridSize_z};
    dim3 block{blockSize_x, blockSize_y, blockSize_z};
    stencil_3d27pt_kernel_1<<<grid, block>>>(input, output);
    stencil_3d27pt_kernel_1<<<grid, block>>>(output, input);
    hipError_t status = hipDeviceSynchronize();
    if (status != hipSuccess) {
        printf("cuda error: %s\n", hipGetErrorString(status));
    }
}